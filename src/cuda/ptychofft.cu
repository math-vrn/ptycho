#include "hip/hip_runtime.h"
#include "ptychofft.cuh"
#include "kernels.cuh"
#include<stdio.h>

ptychofft::ptychofft(size_t Ntheta_, size_t Nz_, size_t N_, size_t Ntheta0,
	size_t Nscan_, size_t detx_, size_t dety_, size_t Nprb_)
{
	N = N_;	
	Ntheta = Ntheta_;
	Nz = Nz_;
	Nscan = Nscan_;
	detx = detx_;
	dety = dety_;
	Nprb = Nprb_;

	hipMalloc((void**)&f,Ntheta*Nz*N*sizeof(float2));
	hipMalloc((void**)&g,Ntheta*Nscan*detx*dety*sizeof(float2));
	hipMalloc((void**)&scanx,Ntheta*Nscan*sizeof(float));
	hipMalloc((void**)&scany,Ntheta*Nscan*sizeof(float));
	hipMalloc((void**)&shiftx,Ntheta*Nscan*sizeof(float2));
	hipMalloc((void**)&shifty,Ntheta*Nscan*sizeof(float2));
	hipMalloc((void**)&prb,Ntheta*Nprb*Nprb*sizeof(float2));
	hipMalloc((void**)&data,Ntheta*Nscan*detx*dety*sizeof(float));	

	int ffts[2];
	int idist;int odist;
	int inembed[2];int onembed[2];
	ffts[0] = detx; ffts[1] = dety;
	idist = detx*dety; odist = detx*dety;
	inembed[0] = detx; inembed[1] = dety;
	onembed[0] = detx; onembed[1] = dety;
	hipfftPlanMany(&plan2dfwd, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Ntheta*Nscan); 
}

ptychofft::~ptychofft()
{	
	hipFree(f);
	hipFree(g);
	hipFree(scanx);
	hipFree(scany);
	hipFree(shiftx);
	hipFree(shifty);
	hipFree(prb);	
	hipFree(data);	
	hipfftDestroy(plan2dfwd);
}

void ptychofft::fwd(size_t g_, size_t f_, size_t prb_, size_t scan_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscan/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	dim3 GS3d1(ceil(detx*dety/(float)BS3d.x),ceil(Nscan/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	dim3 BS2d(32,32);
	dim3 GS2d0(ceil(Nscan/(float)BS2d.x),ceil(Ntheta/(float)BS2d.y));
	
	hipMemcpy(f,(float2*)f_,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);
	hipMemset(g,0,Ntheta*Nscan*detx*dety*sizeof(float2));
	hipMemcpy(scanx,&((float*)scan_)[0],Ntheta*Nscan*sizeof(float),hipMemcpyDefault);  	
	hipMemcpy(scany,&((float*)scan_)[Ntheta*Nscan],Ntheta*Nscan*sizeof(float),hipMemcpyDefault);  	
	hipMemcpy(prb,(float2*)prb_,Ntheta*Nprb*Nprb*sizeof(float2),hipMemcpyDefault);	
	mul<<<GS3d0,BS3d>>>(g,f,prb,scanx,scany,Ntheta,Nz,N,Nscan,Nprb,detx,dety);
	
	hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_FORWARD);
	takeshifts<<<GS2d0,BS2d>>>(shiftx,shifty,scanx,scany,Ntheta,Nscan);		
	shifts<<<GS3d1,BS3d>>>(g, shiftx, shifty, Ntheta, Nscan, detx*dety);
	hipMemcpy((float2*)g_,g,Ntheta*Nscan*detx*dety*sizeof(float2),hipMemcpyDefault);  	
}

void ptychofft::adj(size_t f_, size_t g_, size_t prb_, size_t scan_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscan/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	dim3 GS3d1(ceil(detx*dety/(float)BS3d.x),ceil(Nscan/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	dim3 BS2d(32,32);
	dim3 GS2d0(ceil(Nscan/(float)BS2d.x),ceil(Ntheta/(float)BS2d.y));
	
	hipMemcpy(g,(float2*)g_,Ntheta*Nscan*detx*dety*sizeof(float2),hipMemcpyDefault);  	
	hipMemset(f,0,Ntheta*Nz*N*sizeof(float2));	
	hipMemcpy(scanx,&((float*)scan_)[0],Ntheta*Nscan*sizeof(float),hipMemcpyDefault);  	
	hipMemcpy(scany,&((float*)scan_)[Ntheta*Nscan],Ntheta*Nscan*sizeof(float),hipMemcpyDefault);  	
	hipMemcpy(prb,(float2*)prb_,Ntheta*Nprb*Nprb*sizeof(float2),hipMemcpyDefault);		

	takeshifts<<<GS2d0,BS2d>>>(shiftx,shifty,scanx,scany,Ntheta,Nscan);		
	shiftsa<<<GS3d1,BS3d>>>(g, shiftx, shifty, Ntheta, Nscan, detx*dety);
	hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_BACKWARD);
	mula<<<GS3d0,BS3d>>>(f,g,prb,scanx,scany,Ntheta,Nz,N,Nscan,Nprb,detx,dety);
	hipMemcpy((float2*)f_,f,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);  	
}






