#include "hip/hip_runtime.h"
#include "ptychofft.cuh"
#include "kernels.cuh"
#include<stdio.h>

ptychofft::ptychofft(size_t Ntheta_, size_t Nz_, size_t N_, size_t Ntheta0,
	size_t Nscan_, size_t detx_, size_t dety_, size_t Nprb_)
{
	N = N_;	
	Ntheta = Ntheta_/1;
	Nz = Nz_;
	Nscan = Nscan_;
	detx = detx_;
	dety = dety_;
	Nprb = Nprb_;

	hipMalloc((void**)&f,Ntheta*Nz*N*sizeof(float2));
	hipMalloc((void**)&g,Ntheta*Nscan*detx*dety*sizeof(float2));
	hipMalloc((void**)&scanx,1*Ntheta*Nscan*sizeof(float));
	hipMalloc((void**)&scany,1*Ntheta*Nscan*sizeof(float));
	hipMalloc((void**)&shiftx,1*Ntheta*Nscan*sizeof(float2));
	hipMalloc((void**)&shifty,1*Ntheta*Nscan*sizeof(float2));
	hipMalloc((void**)&prb,Nprb*Nprb*sizeof(float2));
	hipMalloc((void**)&data,Ntheta*Nscan*detx*dety*sizeof(float));	

	int ffts[2];
	int idist;int odist;
	int inembed[2];int onembed[2];
	ffts[0] = detx; ffts[1] = dety;
	idist = detx*dety; odist = detx*dety;
	inembed[0] = detx; inembed[1] = dety;
	onembed[0] = detx; onembed[1] = dety;
	hipfftPlanMany(&plan2dfwd, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Ntheta*Nscan); 
}

ptychofft::~ptychofft()
{	
	hipFree(f);
	hipFree(g);
	hipFree(scanx);
	hipFree(scany);
	hipFree(shiftx);
	hipFree(shifty);
	hipFree(prb);	
	hipFree(data);	
	hipfftDestroy(plan2dfwd);
}

void ptychofft::setobj(size_t scan_, size_t prb_)
{
	hipMemcpy(scanx,&((float*)scan_)[0],1*Ntheta*Nscan*sizeof(float),hipMemcpyDefault);  	
	hipMemcpy(scany,&((float*)scan_)[1*Ntheta*Nscan],1*Ntheta*Nscan*sizeof(float),hipMemcpyDefault);  	
	hipMemcpy(prb,(float2*)prb_,Nprb*Nprb*sizeof(float2),hipMemcpyDefault);
	dim3 BS2d(32,32);
	dim3 GS2d0(ceil(Nscan/(float)BS2d.x),ceil(1*Ntheta/(float)BS2d.y));
	takeshifts<<<GS2d0,BS2d>>>(shiftx,shifty,scanx,scany,1*Ntheta,Nscan);	
}

void ptychofft::fwd(size_t g_, size_t f_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscan/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	dim3 GS3d1(ceil(detx*dety/(float)BS3d.x),ceil(Nscan/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	
	for(int i=0;i<1;i++)	
	{
		hipMemcpy(f,&((float2*)f_)[i*Ntheta*Nz*N],Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);
		hipMemset(g,0,Ntheta*Nscan*detx*dety*sizeof(float2));
		
		
		mul<<<GS3d0,BS3d>>>(g,f,prb,&scanx[i*Ntheta*Nscan],&scany[i*Ntheta*Nscan],Ntheta,Nz,N,Nscan,Nprb,detx,dety);
		
		hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_FORWARD);
		shifts<<<GS3d1,BS3d>>>(g, &shiftx[i*Ntheta*Nscan], &shifty[i*Ntheta*Nscan], Ntheta, Nscan, detx*dety);
		hipMemcpy(&((float2*)g_)[i*Ntheta*Nscan*detx*dety],g,Ntheta*Nscan*detx*dety*sizeof(float2),hipMemcpyDefault);  	
	}
}

void ptychofft::adj(size_t f_, size_t g_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscan/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	dim3 GS3d1(ceil(detx*dety/(float)BS3d.x),ceil(Nscan/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	
	for(int i=0;i<1;i++)	
	{
		hipMemcpy(g,&((float2*)g_)[i*Ntheta*Nscan*detx*dety],Ntheta*Nscan*detx*dety*sizeof(float2),hipMemcpyDefault);  	
		hipMemset(f,0,Ntheta*Nz*N*sizeof(float2));	
		shiftsa<<<GS3d1,BS3d>>>(g, &shiftx[i*Ntheta*Nscan], &shifty[i*Ntheta*Nscan], Ntheta, Nscan, detx*dety);
		hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_BACKWARD);
		mula<<<GS3d0,BS3d>>>(f,g,prb,&scanx[i*Ntheta*Nscan],&scany[i*Ntheta*Nscan],Ntheta,Nz,N,Nscan,Nprb,detx,dety);
		hipMemcpy(&((float2*)f_)[i*Ntheta*Nz*N],f,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);  	
	}
}






