#include "hip/hip_runtime.h"
#include "radonusfft.cuh"
#include "kernels.cuh"
#include <stdio.h>

radonusfft::radonusfft(size_t Ntheta_, size_t Nz_, size_t N_)
{
	N = N_*3/2;
	Ntheta = Ntheta_;
	Nz = Nz_;
	float eps = 1e-6;
	mu = -log(eps)/(2*N*N);
	M = ceil(2*N*1/PI*sqrt(-mu*log(eps)+(mu*N)*(mu*N)/4));

	hipMalloc((void**)&f,N*N*Nz*sizeof(float2));
	hipMalloc((void**)&g,N*Ntheta*Nz*sizeof(float2));
	hipMalloc((void**)&fde,2*N*2*N*Nz*sizeof(float2));
	hipMalloc((void**)&fdee,(2*N+2*M)*(2*N+2*M)*Nz*sizeof(float2));

	hipMalloc((void**)&x,N*Ntheta*sizeof(float));
	hipMalloc((void**)&y,N*Ntheta*sizeof(float));
	hipMalloc((void**)&theta,Ntheta*sizeof(float));

	int ffts[2];
	int idist;int odist;
	int inembed[2];int onembed[2];
	//fft 2d 
	ffts[0] = 2*N; ffts[1] = 2*N;
	idist = 2*N*2*N;odist = (2*N+2*M)*(2*N+2*M);
	inembed[0] = 2*N; inembed[1] = 2*N;
	onembed[0] = 2*N+2*M; onembed[1] = 2*N+2*M;
	hipfftPlanMany(&plan2dfwd, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Nz); 
	hipfftPlanMany(&plan2dadj, 2, ffts, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2C, Nz); 

	//fft 1d	
	ffts[0] = N;
	idist = N;odist = N;
	inembed[0] = N;onembed[0] = N;
	hipfftPlanMany(&plan1d, 1, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Ntheta*Nz);
}

radonusfft::~radonusfft()
{	
	hipFree(f);
	hipFree(g);			
	hipFree(fde);
	hipFree(fdee);
	hipFree(x);
	hipFree(y);
	hipfftDestroy(plan2dfwd);
	hipfftDestroy(plan2dadj);
	hipfftDestroy(plan1d);
}

void radonusfft::fwd(size_t g_, size_t f_)
{	
	dim3 BS2d(32,32);
	dim3 BS3d(32,32,1);

	dim3 GS2d0(ceil(N/(float)BS2d.x),ceil(Ntheta/(float)BS2d.y));
	dim3 GS3d0(ceil(N/(float)BS3d.x),ceil(N/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d1(ceil(2*N/(float)BS3d.x),ceil(2*N/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d2(ceil((2*N+2*M)/(float)BS3d.x),ceil((2*N+2*M)/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d3(ceil(N/(float)BS3d.x),ceil(Ntheta/(float)BS3d.y),ceil(Nz/(float)BS3d.z));

	//hipMemcpy(f,(float2*)f_,N*N*Nz*sizeof(float2),hipMemcpyDefault);
	//padded version
	hipMemset(f,0,N*N*Nz*sizeof(float2));
	for(int iz=0;iz<Nz;iz++)
		hipMemcpy2D(&f[iz*N*N+N/6*N+N/6],N*sizeof(float2),&((float2*)f_)[iz*N/3*2*N/3*2],N/3*2*sizeof(float2),N/3*2*sizeof(float2),N/3*2*1,hipMemcpyDefault);

	hipMemset(fde,0,2*N*2*N*Nz*sizeof(float2));
	hipMemset(fdee,0,(2*N+2*M)*(2*N+2*M)*Nz*sizeof(float2));

	pad2<<<GS3d0, BS3d>>>(f,N,Nz);
	circ<<<GS3d0, BS3d>>>(f,1.0f/N,N,Nz);
	takexy<<<GS2d0, BS2d>>>(x,y,theta,N,Ntheta);

	divphi<<<GS3d0, BS3d>>>(fde,f,mu,N,Nz);
	fftshiftc<<<GS3d1, BS3d>>>(fde,2*N,Nz);
	hipfftExecC2C(plan2dfwd, (hipfftComplex*)fde,(hipfftComplex*)&fdee[M+M*(2*N+2*M)],HIPFFT_FORWARD);
	fftshiftc<<<GS3d2, BS3d>>>(fdee,2*N+2*M,Nz);

	wrap<<<GS3d2, BS3d>>>(fdee,N,Nz,M);
	gather<<<GS3d3, BS3d>>>(g,fdee,x,y,M,mu,N,Ntheta,Nz);

	fftshift1c<<<GS3d3, BS3d>>>(g,N,Ntheta,Nz);
	hipfftExecC2C(plan1d, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_BACKWARD);
	fftshift1c<<<GS3d3, BS3d>>>(g,N,Ntheta,Nz);

	//hipMemcpy((float2*)g_,g,N*Ntheta*Nz*sizeof(float2),hipMemcpyDefault);
	hipMemcpy2D((float2*)g_,N/3*2*sizeof(float2),&g[N/6],N*sizeof(float2),N/3*2*sizeof(float2),Ntheta*Nz,hipMemcpyDefault);  	
}

void radonusfft::adj(size_t f_, size_t g_)
{
	dim3 BS2d(32,32);
	dim3 BS3d(32,32,1);

	dim3 GS2d0(ceil(N/(float)BS2d.x),ceil(Ntheta/(float)BS2d.y));
	dim3 GS3d0(ceil(N/(float)BS3d.x),ceil(N/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d1(ceil(2*N/(float)BS3d.x),ceil(2*N/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d2(ceil((2*N+2*M)/(float)BS3d.x),ceil((2*N+2*M)/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d3(ceil(N/(float)BS3d.x),ceil(Ntheta/(float)BS3d.y),ceil(Nz/(float)BS3d.z));

	//hipMemcpy(g,(float2*)g_,N*Ntheta*Nz*sizeof(float2),hipMemcpyDefault);
	hipMemset(g,0,N*Ntheta*Nz*sizeof(float2));
	hipMemcpy2D(&g[N/6],N*sizeof(float2),(float2*)g_,N/3*2*sizeof(float2),N/3*2*sizeof(float2),Ntheta*Nz,hipMemcpyDefault);  	


	hipMemset(fde,0,(2*N+2*M)*(2*N+2*M)*Nz*sizeof(float2));
	hipMemset(fdee,0,(2*N+2*M)*(2*N+2*M)*Nz*sizeof(float2));


	//padded version
	pad<<<GS3d3,BS3d>>>(g,N,Ntheta,Nz);
	takexy<<<GS2d0, BS2d>>>(x,y,theta,N,Ntheta);

	fftshift1c<<<GS3d3, BS3d>>>(g,N,Ntheta,Nz);
	hipfftExecC2C(plan1d, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_FORWARD);
	fftshift1c<<<GS3d3, BS3d>>>(g,N,Ntheta,Nz);
	//applyfilter<<<GS3d3, BS3d>>>(g,N,Ntheta,Nz);


	scatter<<<GS3d3, BS3d>>>(fdee,g,x,y,M,mu,N,Ntheta,Nz);
	wrapadj<<<GS3d2, BS3d>>>(fdee,N,Nz,M);

	fftshiftc<<<GS3d2, BS3d>>>(fdee,2*N+2*M,Nz);
	hipfftExecC2C(plan2dadj, (hipfftComplex*)&fdee[M+M*(2*N+2*M)],(hipfftComplex*)fde,HIPFFT_BACKWARD);
	fftshiftc<<<GS3d1, BS3d>>>(fde,2*N,Nz);

	unpaddivphi<<<GS3d0, BS3d>>>(f,fde,mu,N,Nz);
	circ<<<GS3d0, BS3d>>>(f,1.0f/N,N,Nz);

	//hipMemcpy((float2*)f_,f,N*N*Nz*sizeof(float2),hipMemcpyDefault);  	
	//padded version
	for(int iz=0;iz<Nz;iz++)
		hipMemcpy2D(&((float2*)f_)[iz*N/3*2*N/3*2],N/3*2*sizeof(float2),&f[iz*N*N+N/6*N+N/6],N*sizeof(float2),N/3*2*sizeof(float2),N/3*2*1,hipMemcpyDefault);

}

void radonusfft::setobj(size_t theta_)
{
	hipMemcpy(theta,(float*)theta_,Ntheta*sizeof(float),hipMemcpyDefault);  	
}
